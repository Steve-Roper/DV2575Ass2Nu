#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#define epsilon 0.00000001

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void Gaussian(vector<vector<double>> &data, int size);
void ForwardElim(vector<vector<double>> &data, int size);
void BackSub(vector<vector<double>> &data, int size);
void SwapRows(vector<vector<double>> &data, int size, int upperRow, int lowerRow);
void CompareResults(vector<vector<double>> &data, vector<vector<double>> &data2, int size);

void GPUGaussian(vector<vector<double>> &data, int size);
__global__ void KernelForwardElim(double* upper, double* lower, int* _size, double* multiplier, int* _upperRow);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

void FillMatrix(int size, vector<vector<double>> &data);

int main()
{
	srand(time(NULL));
    // Add vectors in parallel.
	hipError_t cudaStatus = hipSuccess;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	int size = 15;
	int nrOfThreads = 1;
	//vector<vector<double>> data = { { 8,5,7 },{ 4,6,3 },{ 3,1,9 } };
	//vector<double> vector = { 2,5,3 };
	vector<vector<double>> data, data2;
	//data.reserve(size * size + 1);
	//data2.reserve(size * size + 1);
	//data2 = data;
	//data2[0].push_back(2);
	//data2[1].push_back(5);
	//data2[2].push_back(3);
	//data = data2;
	FillMatrix(size, data);
	data2 = data;
	Gaussian(data, size);
	GPUGaussian(data2, size);
	CompareResults(data, data2, size);
	
	// cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


	cin.get();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

void Gaussian(vector<vector<double>> &data, int size)
{
	ForwardElim(data, size);
	BackSub(data, size);
}

void ForwardElim(vector<vector<double>> &data, int size)
{
	for (unsigned int i = 0; i < size - 1; ++i)
	{
		if (abs(data[i][i]) < epsilon)
		{
			int j = NULL;
			for (j = i + 1; j < size; ++j)
			{
				if (abs(data[j][i]) > epsilon)
				{
					SwapRows(data, size, i, j);
					break;
				}
			}

			if (j == size - 1)
				data[i][i] = 1;
		}
		double upper = data[i][i];
		for (unsigned int j = i + 1; j < size; ++j)
		{
			bool breaK = false;
			while (j < size && abs(data[j][i]) < epsilon)
			{
				++j;
				if (j == size)
				{
					breaK = true;
					break;
				}
			}
			if (breaK)
			{
				break;
			}
			double lower = data[j][i];
			double multiplier = upper / lower;
			for (unsigned int k = 0; k < size + 1; ++k)
			{
				data[j][k] *= multiplier;
				data[j][k] -= data[i][k];
			}
		}
	}
}

void BackSub(vector<vector<double>> &data, int size)
{
	for (int i = size - 1; i >= 0; --i)
	{
		data[i][size] /= data[i][i];
		data[i][i] = 1;
		for (int j = i - 1; j >= 0; --j) //s�tter j = 2 f�rst och sen s� l�nge 2 > 3 what
		{
			double subtrahend = data[j][i] * data[i][size];
			data[j][size] -= subtrahend;
			data[j][i] = 0;
		}
	}
}

void GPUGaussian(vector<vector<double>>& data, int size)
{
	double* devUpperRow		= 0;
	double* devLowerRow		= 0;
	int* devSize			= 0;
	double* devMultiplier	= 0;
	int* devUpperRowIdx		= 0;

	//malloc rows
	hipError_t cudaStatus = hipMalloc((void**)&devUpperRow, (size + 1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for upperRow\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devLowerRow, (size + 1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for lowerRow\n");
		return;
	}
	//malloc rest
	cudaStatus = hipMalloc((void**)&devSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for size\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devMultiplier, sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for multiplier\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devUpperRowIdx, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for upperRowIdx\n");
		return;
	}

	//memcpy size
	cudaStatus = hipMemcpy(devSize, &size, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for size\n");
		return;
	}

	for (int i = 0; i < (size - 1); ++i)
	{
		if (abs(data[i][i]) < epsilon)
		{
			int j = NULL;
			for (j = i + 1; j < size; ++j)
			{
				if (abs(data[j][i]) > epsilon)
				{
					SwapRows(data, size, i, j);
					break;
				}
			}

			if (j == size - 1)
				data[i][i] = 1;
		}

		double* tempData = data[i].data();
		cudaStatus = hipMemcpy((void*)devUpperRow, (void*)tempData, (size + 1) * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for upperRow\n");
			return;
		}
		cudaStatus = hipMemcpy((void*)devUpperRowIdx, (void*)&i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for multiplier\n");
			return;
		}

		//for a given pivot element, reduce all items below to zero
		for (int j = i + 1; j < size; ++j)
		{
			bool breaK = false;
			while (abs(data[j][i]) < epsilon)
			{
				++j;
				if (j == size)
				{
					breaK = true;
					break;
				}
			}
			if (breaK)
			{
				break;
			}
			double multiplier = data[i][i] / data[j][i];
			tempData = data[j].data();
			cudaStatus = hipMemcpy((void*)devLowerRow, (void*)tempData, (size + 1) * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for lowerRow HtD\n");
				return;
			}
			cudaStatus = hipMemcpy((void*)devMultiplier, (void*)&multiplier, sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for multiplier\n");
				return;
			}

			KernelForwardElim<<<1, size + 1>>>(devUpperRow, devLowerRow, devSize, devMultiplier, devUpperRowIdx);

			double* lowerRow = (double*)malloc((size + 1) * sizeof(double));

			cudaStatus = hipMemcpy((void*)lowerRow, (void*)devLowerRow, (size + 1) * sizeof(double), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for lowerRow DtH\n");
				return;
			}

			for (int k = 0; k < size + 1; ++k)
			{
				data[j][k] = lowerRow[k];
			}
		}
	}
	BackSub(data, size);
}

void SwapRows(vector<vector<double>> &data, int size, int upperRow, int lowerRow)
{
	vector<double> temp;
	temp = data[upperRow];
	data[upperRow] = data[lowerRow];
	data[lowerRow] = temp;
}

void CompareResults(vector<vector<double>>& data, vector<vector<double>>& data2, int size)
{
	bool test = true;
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size + 1; ++j)
		{
			if (abs(data[i][j] - data2[i][j]) > epsilon)
			{
				cout << "Something went wrong" << endl;
				cout << "CPU: " << data[i][j] << "|\t" << data2[i][j] << endl;
				test = false;
			}
		}
	}
	if (test)
	{
		cout << "CPU and GPU results match!" << endl;
	}
}

//											data[i],	data[j],	data.size(), upperMainDiag/theOneUnderIt 
__global__ void KernelForwardElim(double* upperRow, double* lowerRow, int* _size, double* multiplier, int* _upperRowIdx)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int upperRowIdx = *_upperRowIdx;
	if (col >= upperRowIdx && col <= *_size)
	{
		lowerRow[col] *= *multiplier;
		lowerRow[col] -= upperRow[col];
	}
}

void FillMatrix(int size, vector<vector<double>> &data)
{
	data.clear();

	for (int i = 0; i < size; ++i)
	{
		vector<double> temp;
		for (int j = 0; j < size + 1; ++j)
		{
			temp.push_back(rand() % 10 + 1);
		}
		data.push_back(temp);
		temp.clear();
	}
}