#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#define epsilon 0.000001

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void Gaussian(vector<vector<double>> &data, int size);
void ForwardElim(vector<vector<double>> &data, int size);
void BackSub(vector<vector<double>> &data, int size);
void SwapRows(vector<vector<double>> &data, int size, int upperRow, int lowerRow);
bool CompareResults(vector<vector<double>> &data, vector<vector<double>> &data2, int size);

void GPUGaussian(vector<vector<double>> &data, int size, int blocks, int colsPerThread);
__global__ void KernelForwardElim(double* upper, double* lower, int* _size, double* multiplier, int* _upperRow, int* colsPerThread);


void FillMatrix(int size, vector<vector<double>> &data);

int main()
{
	srand(time(NULL));
    // Add vectors in parallel.
	hipError_t cudaStatus = hipSuccess;

	//int size = 256;
	//int colsPerThread = 1;
	vector<vector<double>> data, data2, backup;
	//double** data, data2, backup;

	for (int size = 128; size < 1025; size *= 2)
	{
		FillMatrix(size, data);
		//backup = (double**)malloc((size + 1) * size * sizeof(double));
		backup = data;
		Gaussian(data, size);
		for (int colsPerThread = 1; colsPerThread < 9; colsPerThread *= 2)
		{
			data2.clear();
			data2 = backup;
			int threads = (size + 1) / colsPerThread;
			int blocks = (threads - 1) / 1024 + 1; /*1024 max for current graphics card used*/
			GPUGaussian(data2, size, blocks, colsPerThread);
			if (!CompareResults(data, data2, size))
			{
				break;
			}
		}
	}
	// cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	cin.get();
    return 0;
}

void Gaussian(vector<vector<double>> &data, int size)
{
	clock_t t;
	t = clock();
	ForwardElim(data, size);
	t = clock() - t;

	std::cout << "CPU Forward Substituion took: " << t << "clicks ("<< ((float)t)/CLOCKS_PER_SEC << " seconds.)" << endl;

	BackSub(data, size);
}

void ForwardElim(vector<vector<double>> &data, int size)
{
	for (unsigned int i = 0; i < size - 1; ++i)
	{
		if (abs(data[i][i]) < epsilon)
		{
			int j = NULL;
			for (j = i + 1; j < size; ++j)
			{
				if (abs(data[j][i]) > epsilon)
				{
					SwapRows(data, size, i, j);
					break;
				}
			}

			if (j == size - 1)
				data[i][i] = 1;
		}
		double upper = data[i][i];
		for (unsigned int j = i + 1; j < size; ++j)
		{
			bool breaK = false;
			while (j < size && abs(data[j][i]) < epsilon)
			{
				++j;
				if (j == size)
				{
					breaK = true;
					break;
				}
			}
			if (breaK)
			{
				break;
			}
			double lower = data[j][i];
			double multiplier = upper / lower;
			for (unsigned int k = i + 1; k < size + 1; ++k)
			{
				data[j][k] *= multiplier;
				data[j][k] -= data[i][k];
			}
		}
	}
}

void BackSub(vector<vector<double>> &data, int size)
{
	for (int i = size - 1; i >= 0; --i)
	{
		data[i][size] /= data[i][i];
		data[i][i] = 1;
		for (int j = i - 1; j >= 0; --j) //s�tter j = 2 f�rst och sen s� l�nge 2 > 3 what
		{
			double subtrahend = data[j][i] * data[i][size];
			data[j][size] -= subtrahend;
			data[j][i] = 0;
		}
	}
}

void GPUGaussian(vector<vector<double>>& data, int size, int blocks, int colsPerThread)
{
	double* devUpperRow		= 0;
	double* devLowerRow		= 0;
	int* devSize			= 0;
	double* devMultiplier	= 0;
	int* devUpperRowIdx		= 0;
	int* devColsPerThread	= 0;

	//malloc rows
	hipError_t cudaStatus = hipMalloc((void**)&devUpperRow, (size + 1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for upperRow\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devLowerRow, (size + 1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for lowerRow\n");
		return;
	}
	//malloc rest
	cudaStatus = hipMalloc((void**)&devSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for size\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devMultiplier, sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for multiplier\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devUpperRowIdx, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for upperRowIdx\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devColsPerThread, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for colsPerThread\n");
		return;
	}

	//memcpy size
	cudaStatus = hipMemcpy(devSize, &size, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for size\n");
		return;
	}
	cudaStatus = hipMemcpy(devColsPerThread, &colsPerThread, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for colsPerThread\n");
		return;
	}

	clock_t t;
	t = clock();
	for (int i = 0; i < (size - 1); ++i)
	{
		if (abs(data[i][i]) < epsilon)
		{
			int j = NULL;
			for (j = i + 1; j < size; ++j)
			{
				if (abs(data[j][i]) > epsilon)
				{
					SwapRows(data, size, i, j);
					break;
				}
			}

			if (j == size - 1)
				data[i][i] = 1;
		}

		double* tempData = data[i].data();
		cudaStatus = hipMemcpy((void*)devUpperRow, (void*)tempData, (size + 1) * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for upperRow\n");
			return;
		}
		cudaStatus = hipMemcpy((void*)devUpperRowIdx, (void*)&i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for multiplier\n");
			return;
		}

		//for a given pivot element, reduce all items below to zero
		for (int j = i + 1; j < size; ++j)
		{
			bool breaK = false;
			while (abs(data[j][i]) < epsilon)
			{
				++j;
				if (j == size)
				{
					breaK = true;
					break;
				}
			}
			if (breaK)
			{
				break;
			}
			double multiplier = data[i][i] / data[j][i];
			tempData = data[j].data();
			cudaStatus = hipMemcpy((void*)devLowerRow, (void*)tempData, (size + 1) * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for lowerRow HtD\n");
				return;
			}
			cudaStatus = hipMemcpy((void*)devMultiplier, (void*)&multiplier, sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for multiplier\n");
				return;
			}

			KernelForwardElim<<<blocks, 1024>>>(devUpperRow, devLowerRow, devSize, devMultiplier, devUpperRowIdx, devColsPerThread);
			

			double* lowerRow = (double*)malloc((size + 1) * sizeof(double));

			cudaStatus = hipMemcpy((void*)lowerRow, (void*)devLowerRow, (size + 1) * sizeof(double), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for lowerRow DtH\n");
				return;
			}

			for (int k = 0; k < size + 1; ++k)
			{
				data[j][k] = lowerRow[k];
			}
		}	
	}
	t = clock() - t;
	std::cout << "GPU Forward Substituion took: " << t << "clicks (" << ((float)t) / CLOCKS_PER_SEC << " seconds.)" << endl;
	BackSub(data, size);

	hipFree(devUpperRow);
	hipFree(devLowerRow);
	hipFree(devSize);
	hipFree(devMultiplier);
	hipFree(devUpperRowIdx);
	hipFree(devColsPerThread);
}

void SwapRows(vector<vector<double>> &data, int size, int upperRow, int lowerRow)
{
	vector<double> temp;
	temp = data[upperRow];
	data[upperRow] = data[lowerRow];
	data[lowerRow] = temp;
}

bool CompareResults(vector<vector<double>>& data, vector<vector<double>>& data2, int size)
{
	bool test = true;
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size + 1; ++j)
		{
			if (abs(data[i][j] - data2[i][j]) > epsilon && abs(data[i][j]) > epsilon && abs(data2[i][j]) > epsilon)
			{
				cout << "Something went wrong" << endl;
				cout << "CPU: " << data[i][j] << "|\tGPU:" << data2[i][j] << endl;
				test = false;
			}
		}
	}
	if (test)
	{
		cout << "CPU and GPU results match!" << endl;
	}
	return test;
}

//											data[i],	data[j],	data.size(), upperMainDiag/theOneUnderIt 
__global__ void KernelForwardElim(double* upperRow, double* lowerRow, int* _size, double* multiplier, int* _upperRowIdx, int* colsPerThread)
{
	int _colsPerThread = *colsPerThread;
	int startCol = (threadIdx.x + blockIdx.x * blockDim.x) * _colsPerThread;
	int upperRowIdx = *_upperRowIdx;
	for (int col = startCol; col < _colsPerThread + startCol; ++col)
	{
		if (col > upperRowIdx && col <= *_size)
		{
			lowerRow[col] *= *multiplier;
			lowerRow[col] -= upperRow[col];
		}
	}
}

void FillMatrix(int size, vector<vector<double>> &data)
{
	data.clear();

	for (int i = 0; i < size; ++i)
	{
		vector<double> temp;
		for (int j = 0; j < size + 1; ++j)
		{
			temp.push_back(rand() % 10 + 1);
		}
		data.push_back(temp);
		temp.clear();
	}
}