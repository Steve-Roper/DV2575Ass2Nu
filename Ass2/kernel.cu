
#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>
#include <vector>
#include <iostream>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void Gaussian(vector<vector<float>> &data, vector<float> &vector);
void ForwardElim(vector<vector<float>> &data, vector<float> &vector);
void BackSub(vector<vector<float>> &data, std::vector<float> &vector);

void GPUGaussian(vector<vector<float>> &data, int size);
__global__ void KernelForwardElim(float* upper, float* lower, int* _size, float* multiplier, int* _upperRow);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

void FillMatrix(int size, vector<vector<float>> &data, vector<float> &vector);

int main()
{
    // Add vectors in parallel.
	hipError_t cudaStatus = hipSuccess;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	int size = 3;
	vector<vector<float>> data = { { 8,5,7 },{ 4,6,3 },{ 3,1,9 } };
	vector<float> vector = { 2,5,3 };
	std::vector<std::vector<float>> data2;
	data2 = data;
	data2[0].push_back(2);
	data2[1].push_back(5);
	data2[2].push_back(3);
	//FillMatrix(3, data, vector);
	Gaussian(data, vector);
	GPUGaussian(data2, size);
	
	// cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

void Gaussian(vector<vector<float>> &data, std::vector<float> &vector)
{
	ForwardElim(data, vector);
	//BackSub(data, vector);
}

void ForwardElim(vector<vector<float>> &data, std::vector<float> &vector)
{
	for (unsigned int i = 0; i < (data.size() - 1); ++i)
	{
		float upper = data[i][i];
		for (unsigned int j = i; j < (data.size() - 1); ++j)
		{
			float lower = data[j + 1][i];
			float multiplier = upper / lower;
			vector[j + 1] *= multiplier;
			vector[j + 1] -= vector[i];
			for (unsigned int k = 0; k < data.size(); ++k)
			{
				data[j + 1][k] *= multiplier;
				data[j + 1][k] -= data[i][k];
			}
		}
	}
}

void BackSub(vector<vector<float>> &data, std::vector<float> &vector)
{
	for (int i = data.size() - 1; i >= 0; --i)
	{
		vector[i] /= data[i][i];
		data[i][i] = 1;
		for (int j = i - 1; j >= 0; --j) //s�tter j = 2 f�rst och sen s� l�nge 2 > 3 what
		{
			float subtrahend = data[j][i] * vector[i];
			vector[j] -= subtrahend;
			data[j][i] = 0;
		}
	}
}

void GPUGaussian(vector<vector<float>>& data, int size)
{
	float* devUpperRow		= 0;
	float* devLowerRow		= 0;
	int* devSize			= 0;
	float* devMultiplier	= 0;
	int* devUpperRowIdx		= 0;

	//malloc rows
	hipError_t cudaStatus = hipMalloc((void**)&devUpperRow, (size + 1) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for upperRow\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devLowerRow, (size + 1) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for lowerRow\n");
		return;
	}
	//malloc rest
	cudaStatus = hipMalloc((void**)&devSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for size\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devMultiplier, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for multiplier\n");
		return;
	}
	cudaStatus = hipMalloc((void**)&devUpperRowIdx, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for upperRowIdx\n");
		return;
	}

	//memcpy size
	cudaStatus = hipMemcpy(devSize, &size, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for size\n");
		return;
	}

	for (int i = 0; i < (size - 1); ++i)
	{
		//for a given pivot row, reduce all items below to zero
		for (int j = i + 1; j < (size - 1); ++j)
		{
			float multiplier = data[i][i] / data[j][i];
			cudaStatus = hipMemcpy(devUpperRow, data[i].data(), (size + 1) * sizeof(float), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for upperRow\n");
				return;
			}
			cudaStatus = hipMemcpy(devLowerRow, data[j].data(), (size + 1) * sizeof(float), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for lowerRow\n");
				return;
			}
			cudaStatus = hipMemcpy(devMultiplier, &multiplier, sizeof(float), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for multiplier\n");
				return;
			}
			cudaStatus = hipMemcpy(devUpperRowIdx, &i, sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for multiplier\n");
				return;
			}

			KernelForwardElim<<<1, 4>>>(devUpperRow, devLowerRow, devSize, devMultiplier, devUpperRowIdx);

			float* upperRow = 0;
			float* lowerRow = 0;

			cudaStatus = hipMemcpy(upperRow, devUpperRow, (size + 1) * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for upperRow\n");
				return;
			}
			cudaStatus = hipMemcpy(lowerRow, devLowerRow, (size + 1) * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for lowerRow\n");
				return;
			}

			int stopper = 0;
		}
	}


}
//											data[i],	data[j],	data.size(), upperMainDiag/theOneUnderIt 
__global__ void KernelForwardElim(float* upperRow, float* lowerRow, int* _size, float* multiplier, int* _upperRowIdx)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int upperRowIdx = *_upperRowIdx;
	if (col >= upperRowIdx)
	{
		int size = *_size;
		lowerRow[col] *= *multiplier;
		lowerRow[col] -= upperRow[col];
	}
}

void FillMatrix(int size, vector<vector<float>> &data, vector<float> &vector)
{
	data.clear();
	vector.clear();

	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			vector.push_back(rand() % 10 + 1); //values between 1 and 10
			data.push_back(vector);
			vector.clear();
			//pushback random values to data[i][j]
		}
	}
	for (int i = 0; i < size; ++i)
	{
		
	}
}